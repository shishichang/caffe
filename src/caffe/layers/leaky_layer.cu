#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/leaky_layer.hpp"


namespace caffe {

//add by yang
template <typename Dtype>
__global__ void LeakyForward(const int n, const Dtype* in, Dtype* out){
    CUDA_KERNEL_LOOP(index, n){
        out[index] = in[index] > 0 ? in[index] : in[index]*0.1;
    }
}

template <typename Dtype>
void LeakyLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  LeakyForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}
INSTANTIATE_LAYER_GPU_FUNCS(LeakyLayer);

}//namespace caffe